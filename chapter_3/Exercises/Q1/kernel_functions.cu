/**
 * Question:
 * Implement a matrix multiplication kernel on same-width square matrices where:
 * a. each thread produces one output matrix row.
 * b. each thread produces one output matrix column.
 * 
 * Please refer to the README.md in the Exercises directory for
 * further descriptions of what I am trying to do in the code.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 
// Thread block size.
#define BLOCK_SIZE 16
 
typedef void (MatrixMultiplicationFuction)(float*, float*, float*, int);
 
 
/**
 * The matmul kernel function where each thread produces one output matrix row.
 */
__global__
void Question1AKernel(
    float* matrix_M,
    float* matrix_N,
    float* matrix_Out,
    int Width
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
 
    if (row < Width) {
        for (int col = 0; col < Width; ++col) {
            int out_value = 0;
            for (int k = 0; k < Width; ++k) {
                out_value += matrix_M[row * Width + k] * matrix_N[Width * k + col];
            }
            matrix_Out[row * Width + col] = out_value;
        }
    }
}
 
 
/**
 * The matmul kernel function where each thread produces one output column row.
 */
__global__
void Question1BKernel(
    float* matrix_M,
    float* matrix_N,
    float* matrix_Out,
    int Width
) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
 
    if (col < Width) {
        for (int row = 0; row < Width; ++row) {
            int out_value = 0;
            for (int k = 0; k < Width; ++k) {
                out_value += matrix_M[row * Width + k] * matrix_N[Width * k + col];
            }
            matrix_Out[row * Width + col] = out_value;
        }
    }
}
 
 
/**
 * The host function, to deal with memory allocations and kernel function calls.
 */
void runMatrixMultiplication(
    float* matrix_M_h,
    float* matrix_N_h,
    float* matrix_Out_h,
    int Width,
    MatrixMultiplicationFuction* matmul_kernel
) {
    printf("Width is %d\n", Width);
    // Get size in bytes.
    size_t size = Width * Width * sizeof(float);
 
    // Load and copy matrix M and N to device memory.
    float * matrix_M_d, * matrix_N_d, * matrix_Out_d;
    hipMalloc((void***)&matrix_M_d, size);
    hipMemcpy(matrix_M_d, matrix_M_h, size, hipMemcpyHostToDevice);
 
    hipMalloc((void***)&matrix_N_d, size);
    hipMemcpy(matrix_N_d, matrix_N_h, size, hipMemcpyHostToDevice);
 
    hipMalloc((void***)&matrix_Out_d, size);
 
    // Invoke kernel.
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(ceil(Width / (BLOCK_SIZE * 1.0)));
 
    // Initialize CUDA events to time the kernel function run.
    // For more information: https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul_kernel<<<dimGrid, dimBlock>>>(matrix_M_d, matrix_N_d, matrix_Out_d, Width);
    hipEventRecord(stop);

    // Copy the output matrix from the device memory.
    hipMemcpy(matrix_Out_h, matrix_Out_d, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float time_spent_ms = 0;
    hipEventElapsedTime(&time_spent_ms, start, stop);
    printf("Total duration: %.7f\n", time_spent_ms / 1000.0);

    // Free device vectors.
    hipFree(matrix_M_d);
    hipFree(matrix_N_d);
    hipFree(matrix_Out_d);
}


void run_kernel(
    float * matrix_M,
    float * matrix_N,
    int Width,
    MatrixMultiplicationFuction* matmul_kernel,
    const char * label
) {
    printf("Running %s...\n", label);

    float * matrix_Out = (float *) malloc(Width * Width * sizeof(float));

    runMatrixMultiplication(matrix_M, matrix_N, matrix_Out, Width, matmul_kernel);
    
    free(matrix_Out);
    return;
}
